#include "hip/hip_runtime.h"
#include <vector_functions.h>

#include "CUDASurface.cuh"
#include "GPUParallel.cuh"

void BlitLinearBufferToSurface(
    float4* buffer,
    hipSurfaceObject_t surface_object,
    int width,
    int height)
{
    GPUParallelFor2D(
        "Copy float4 to surface object",
        make_int2(width, height),
        GPU_LAMBDA_Ex(int i, int j) {
            float4 val = buffer[i * width + j];
            surf2Dwrite(val, surface_object, sizeof(float4) * j, i);
        });
}

void BlitLinearBufferToSurface(
    float3* buffer,
    hipSurfaceObject_t surface_object,
    int width,
    int height)
{
    GPUParallelFor2D(
        "Copy float3 to surface object",
        make_int2(width, height),
        GPU_LAMBDA_Ex(int i, int j) {
            auto val_3 = buffer[i * width + j];
            float4 val = make_float4(val_3.x, val_3.y, val_3.z, 1);

            surf2Dwrite(val, surface_object, sizeof(float4) * j, i);
        });
}

void ComposeChannels(
    float4* target,
    float* x,
    float* y,
    float* z,
    float* w,
    int size)
{
    GPUParallelFor(
        "Compose 4 channels into float4", size, GPU_LAMBDA_Ex(int i) {
            target[i] = make_float4(x[i], y[i], z[i], w[i]);
        });
}

void ComposeChannels(float4* target, float* x, float* y, float* z, int size)
{
    GPUParallelFor(
        "Compose 3 channels into float4", size, GPU_LAMBDA_Ex(int i) {
            target[i] = make_float4(x[i], y[i], z[i], 1);
        });
}

void ComposeChannels(float3* target, float* x, float* y, float* z, int size)
{
    GPUParallelFor(
        "Compose 4 channels into float3", size, GPU_LAMBDA_Ex(int i) {
            target[i] = make_float3(x[i], y[i], z[i]);
        });
}
